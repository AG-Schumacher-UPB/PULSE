#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <complex>
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda_complex.cuh"
#include "kernel_runge_function.cuh"
#include "kernel_summation.cuh"
#include "kernel_ringstate.cuh"
#include "kernel_fft.cuh"
#include "system.hpp"
#include "kernel.hpp"

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

/*
TODOs for Optimization:
- remove shift kernel and combine it in mask
- remove index to row/col calculation, instead use map index->row and index->col to avoid expensive % modulo operator
- cache pump and pulse shapes
- cache fft mask -> shift mask array
- calculate everything in one big kernel; no need for intermediate arrays and multiple kernel calls
*/


/*
* Helper variable for caching the current time for FFT evaluations.
* We dont need this variable anywhere else, so we just create it 
* locally to this file here.
*/
double cached_t = 0.0;

/*
* This function iterates the Runge Kutta Kernel using a fixed time step
* @param system The system to iterate
* @param evaluate_pulse If true, the pulse is evaluated at the current time step
*/
void iterateFixedTimestepRungeKutta(System& system, bool evaluate_pulse, dim3 block_size, dim3 grid_size) {
    // Iterate the Runge Function on the current Psi and Calculate K1
    rungeFuncKernel<<<grid_size, block_size>>>( system.t, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K1" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K1 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSum<<<grid_size, block_size>>>( 0.5, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K1)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K2
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + 0.5*system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K2" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K2 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSum<<<grid_size, block_size>>>( 0.5, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K2)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K3
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + 0.5*system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K3" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K3 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSum<<<grid_size, block_size>>>( 1.0, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K3)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K4
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K4" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Calculate the final Runge Kutta sum, saving the result in dev_in_Psi
    rungeFuncSumToFinalFixed<<<grid_size, block_size>>>( dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus );
    CHECK_CUDA_ERROR( {}, "Final Sum" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
}

void iterateVariableTimestepRungeKutta(System& system, bool evaluate_pulse, dim3 block_size, dim3 grid_size) {
    bool accept = false;
    do {
    // Iterate the Runge Function on the current Psi and Calculate K1
    rungeFuncKernel<<<grid_size, block_size>>>( system.t, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K1" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K1 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSumToK2<<<grid_size, block_size>>>( dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K1)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K2
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + RKCoefficients::a2 * system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K2" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K2 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSumToK3<<<grid_size, block_size>>>( dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K1,K2)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K3
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + RKCoefficients::a3 * system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K3" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K3 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSumToK4<<<grid_size, block_size>>>( dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K1,K2,K3)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K4
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + RKCoefficients::a4 * system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K4" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K4 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSumToK5<<<grid_size, block_size>>>( dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K1,K2,K3,K4)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K4
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + RKCoefficients::a5 * system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k5_Psi_Plus, dev_k5_Psi_Minus, dev_k5_n_Plus, dev_k5_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K5" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Sum K5 to get next_Psi_Plus, next_Psi_Minus, next_n_Plus, next_n_Minus
    rungeFuncSumToK6<<<grid_size, block_size>>>( dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus, dev_k5_Psi_Plus, dev_k5_Psi_Minus, dev_k5_n_Plus, dev_k5_n_Minus );
    CHECK_CUDA_ERROR( {}, "Sum(K1,K2,K3,K4,K5)" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Iterate the Runge Function on next_Psi and Calculate K6
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + RKCoefficients::a6 * system.dt, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k6_Psi_Plus, dev_k6_Psi_Minus, dev_k6_n_Plus, dev_k6_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K6" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Calculate the final Runge Kutta sum , saving the result in dev_in_Psi
    rungeFuncSumToFinal<<<grid_size, block_size>>>( dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus, dev_k5_Psi_Plus, dev_k5_Psi_Minus, dev_k5_n_Plus, dev_k5_n_Minus, dev_k6_Psi_Plus, dev_k6_Psi_Minus, dev_k6_n_Plus, dev_k6_n_Minus );
    CHECK_CUDA_ERROR( {}, "Final Sum First" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Calculate the Error Contribution Matrix K7 from dev_in_Psi
    rungeFuncKernel<<<grid_size, block_size>>>( system.t + RKCoefficients::a7 * system.dt, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k7_Psi_Plus, dev_k7_Psi_Minus, dev_k7_n_Plus, dev_k7_n_Minus, dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pump_pol, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_m, dev_pulse_pol, dev_pulse_width, dev_pulse_X, dev_pulse_Y, evaluate_pulse );
    CHECK_CUDA_ERROR( {}, "K7" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );

    // Calculate the Runge Kutta Error. Since we dont need it here anymore, we use one of the K2 cache arrays to do this.
    rungeFuncFinalError<<<grid_size, block_size>>>( dev_rk_error, dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus, dev_k5_Psi_Plus, dev_k5_Psi_Minus, dev_k5_n_Plus, dev_k5_n_Minus, dev_k6_Psi_Plus, dev_k6_Psi_Minus, dev_k6_n_Plus, dev_k6_n_Minus, dev_k7_Psi_Plus, dev_k7_Psi_Minus, dev_k7_n_Plus, dev_k7_n_Minus );
    CHECK_CUDA_ERROR( {}, "Final Sum Error" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    //sum <<<1, system.s_N*system.s_N / 2 >>>(dev_k2_Psi_Minus);


    // Use thrust::reduce to calculate the sum of the error matrix
    double final_error = thrust::reduce(thrust::device, dev_rk_error, dev_rk_error + system.s_N*system.s_N, 0.0, thrust::plus<double>()); 
    
    // Calculate dh
    double dh = pow(system.tolerance / 2. / max(final_error, 1E-15), 0.25);
    // Check if dh is nan
    if (std::isnan(dh)) {
        dh = 1.0;
    }
    if (std::isnan(final_error))
        dh = 0.5;
    //std::cout << " t = " << system.t << " dt = " << system.dt << ", current dh is " << dh << ", new dt would be " << system.dt*dh << ", error is " << final_error << ", which should be less than " << system.tolerance << ", accept? " << (final_error < system.tolerance) << std::endl;
    // Set new timestep
    //system.dt = min(system.dt * dh, system.dt_max);
    if (dh < 1.0)
       system.dt = max(system.dt - system.dt_min*std::floor( 1.0 / dh ), system.dt_min);
    else
       system.dt = min(system.dt + system.dt_min*std::floor( dh ), system.dt_max);
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_s_dt), &system.dt, sizeof( double ) ), "hipMemcpyToSymbol dt" );
    // Accept step if error is below tolerance
    if (final_error < system.tolerance) {
        accept = true;
        // Copy current_Psi to next_Psi
        swap_symbol(dev_current_Psi_Minus, dev_next_Psi_Minus);
        swap_symbol(dev_current_Psi_Plus, dev_next_Psi_Plus);
        swap_symbol(dev_current_n_Minus, dev_next_n_Minus);
        swap_symbol(dev_current_n_Plus, dev_next_n_Plus);
    }
    }
    while ( !accept );
}

/*
 * This function calculates the Fast Fourier Transformation of Psi+ and Psi-
 * and saves the result in dev_fft_plus and dev_fft_minus. These values can
 * then be grabbed using the getDeviceArrays() function. The FFT is then
 * shifted such that k = 0 is in the center of the FFT matrix. Then, the
 * FFT Filter is applied to the FFT, and the FFT is shifted back. Finally,
 * the inverse FFT is calculated and the result is saved in dev_current_Psi_Plus
 * and dev_current_Psi_Minus. The FFT Arrays are shifted once again for
 * visualization purposes.
 * NOTE/OPTIMIZATION: The Shift->Filter->Shift function will be changed later
 * to a cached filter mask, which itself will be shifted.
*/
void calculateFFT(System& system, dim3 block_size, dim3 grid_size){
    CHECK_CUDA_ERROR( hipfftExecZ2Z( plan, (hipfftDoubleComplex*)dev_current_Psi_Plus, (hipfftDoubleComplex*)dev_fft_plus, HIPFFT_FORWARD ), "FFT Exec" );
    CHECK_CUDA_ERROR( hipfftExecZ2Z( plan, (hipfftDoubleComplex*)dev_current_Psi_Minus, (hipfftDoubleComplex*)dev_fft_minus, HIPFFT_FORWARD ), "FFT Exec" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    fftshift_2D<<<grid_size, block_size>>>( dev_fft_plus, dev_fft_minus, system.s_N / 2 );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    CHECK_CUDA_ERROR( {}, "FFT Shift" );
    kernel_maskFFT<<<grid_size, block_size>>>( dev_fft_plus, dev_fft_minus, system.fft_power, system.fft_mask_area, false );
    CHECK_CUDA_ERROR( {}, "FFT Filter" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    fftshift_2D<<<grid_size, block_size>>>( dev_fft_plus, dev_fft_minus, system.s_N / 2 );
    CHECK_CUDA_ERROR( {}, "FFT Shift" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    //// Transform back.
    CHECK_CUDA_ERROR( hipfftExecZ2Z( plan, dev_fft_plus, dev_current_Psi_Plus, HIPFFT_BACKWARD ), "iFFT Exec" );
    CHECK_CUDA_ERROR( hipfftExecZ2Z( plan, dev_fft_minus, dev_current_Psi_Minus, HIPFFT_BACKWARD ), "iFFT Exec" );
    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    // Shift FFT Once again for visualization
    fftshift_2D<<<grid_size, block_size>>>( dev_fft_plus, dev_fft_minus, system.s_N / 2 );
    CHECK_CUDA_ERROR( {}, "FFT Shift" );
}

/**
 * Iterates the Runge-Kutta-Method on the GPU
 * Note, that all device arrays and variables have to be initialized at this point
 * @param evaluate_pulse If true, the pulse is evaluated at the current time step
 * @param t Current time, will be updated to t + dt
 * @param dt Time step, will be updated to the next time step
 * @param s_N Number of grid points in one dimension
 */
void rungeFunctionIterate( System& system, bool evaluate_pulse ) {
    dim3 block_size( 16, 16 );
    dim3 grid_size( ( system.s_N + block_size.x ) / block_size.x, ( system.s_N + block_size.y ) / block_size.y );

    if (system.fixed_time_step)
        iterateFixedTimestepRungeKutta(system, evaluate_pulse, block_size, grid_size);
    else
        iterateVariableTimestepRungeKutta(system, evaluate_pulse, block_size, grid_size);

    // Increase t.
    system.t = system.t + system.dt;

    // For statistical purposes, increase the iteration counter
    system.iteration++;

    // Test: Calculate the FFT of dev_current_Psi_Plus using cufft
    if ( system.t - cached_t < system.fft_every )
        return;
    cached_t = system.t;
    calculateFFT(system, block_size, grid_size);   
}