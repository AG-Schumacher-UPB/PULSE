#include "hip/hip_runtime.h"
#include "kernel_summation.cuh"

/**
 * Summation of the k2 array
 *
 */
__global__ void rungeFuncSumToK2( hipDoubleComplex* __restrict__ out_Psi_Plus, hipDoubleComplex* __restrict__ out_Psi_Minus, hipDoubleComplex* __restrict__ out_n_Plus, hipDoubleComplex* __restrict__ out_n_Minus, hipDoubleComplex* __restrict__ in_Psi_Plus, hipDoubleComplex* __restrict__ in_Psi_Minus, hipDoubleComplex* __restrict__ in_n_Plus, hipDoubleComplex* __restrict__ in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + RKCoefficients::b11 * dev_s_dt * k1_Psi_Plus[i];
        out_Psi_Minus[i] = in_Psi_Minus[i] + RKCoefficients::b11 * dev_s_dt * k1_Psi_Minus[i];
        out_n_Plus[i] = in_n_Plus[i] + RKCoefficients::b11 * dev_s_dt * k1_n_Plus[i];
        out_n_Minus[i] = in_n_Minus[i] + RKCoefficients::b11 * dev_s_dt * k1_n_Minus[i];
    }
}
/*
 * Summation of the k3 array
 */
__global__ void rungeFuncSumToK3( hipDoubleComplex* __restrict__ out_Psi_Plus, hipDoubleComplex* __restrict__ out_Psi_Minus, hipDoubleComplex* __restrict__ out_n_Plus, hipDoubleComplex* __restrict__ out_n_Minus, hipDoubleComplex* __restrict__ in_Psi_Plus, hipDoubleComplex* __restrict__ in_Psi_Minus, hipDoubleComplex* __restrict__ in_n_Plus, hipDoubleComplex* __restrict__ in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k2_Psi_Plus, hipDoubleComplex* __restrict__ k2_Psi_Minus, hipDoubleComplex* __restrict__ k2_n_Plus, hipDoubleComplex* __restrict__ k2_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + RKCoefficients::b21 * dev_s_dt * k1_Psi_Plus[i] + RKCoefficients::b22 * dev_s_dt * k2_Psi_Plus[i];
        out_Psi_Minus[i] = in_Psi_Minus[i] + RKCoefficients::b21 * dev_s_dt * k1_Psi_Minus[i] + RKCoefficients::b22 * dev_s_dt * k2_Psi_Minus[i];
        out_n_Plus[i] = in_n_Plus[i] + RKCoefficients::b21 * dev_s_dt * k1_n_Plus[i] + RKCoefficients::b22 * dev_s_dt * k2_n_Plus[i];
        out_n_Minus[i] = in_n_Minus[i] + RKCoefficients::b21 * dev_s_dt * k1_n_Minus[i] + RKCoefficients::b22 * dev_s_dt * k2_n_Minus[i];
    }
}
/*
 * Summation of the k4 array
 */
__global__ void rungeFuncSumToK4( hipDoubleComplex* __restrict__ out_Psi_Plus, hipDoubleComplex* __restrict__ out_Psi_Minus, hipDoubleComplex* __restrict__ out_n_Plus, hipDoubleComplex* __restrict__ out_n_Minus, hipDoubleComplex* __restrict__ in_Psi_Plus, hipDoubleComplex* __restrict__ in_Psi_Minus, hipDoubleComplex* __restrict__ in_n_Plus, hipDoubleComplex* __restrict__ in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k2_Psi_Plus, hipDoubleComplex* __restrict__ k2_Psi_Minus, hipDoubleComplex* __restrict__ k2_n_Plus, hipDoubleComplex* __restrict__ k2_n_Minus, hipDoubleComplex* __restrict__ k3_Psi_Plus, hipDoubleComplex* __restrict__ k3_Psi_Minus, hipDoubleComplex* __restrict__ k3_n_Plus, hipDoubleComplex* __restrict__ k3_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + RKCoefficients::b31 * dev_s_dt * k1_Psi_Plus[i] + RKCoefficients::b32 * dev_s_dt * k2_Psi_Plus[i] + RKCoefficients::b33 * dev_s_dt * k3_Psi_Plus[i];
        out_Psi_Minus[i] = in_Psi_Minus[i] + RKCoefficients::b31 * dev_s_dt * k1_Psi_Minus[i] + RKCoefficients::b32 * dev_s_dt * k2_Psi_Minus[i] + RKCoefficients::b33 * dev_s_dt * k3_Psi_Minus[i];
        out_n_Plus[i] = in_n_Plus[i] + RKCoefficients::b31 * dev_s_dt * k1_n_Plus[i] + RKCoefficients::b32 * dev_s_dt * k2_n_Plus[i] + RKCoefficients::b33 * dev_s_dt * k3_n_Plus[i];
        out_n_Minus[i] = in_n_Minus[i] + RKCoefficients::b31 * dev_s_dt * k1_n_Minus[i] + RKCoefficients::b32 * dev_s_dt * k2_n_Minus[i] + RKCoefficients::b33 * dev_s_dt * k3_n_Minus[i];
    }
}
/*
 * Summation of the k5 array
 */
__global__ void rungeFuncSumToK5( hipDoubleComplex* __restrict__ out_Psi_Plus, hipDoubleComplex* __restrict__ out_Psi_Minus, hipDoubleComplex* __restrict__ out_n_Plus, hipDoubleComplex* __restrict__ out_n_Minus, hipDoubleComplex* __restrict__ in_Psi_Plus, hipDoubleComplex* __restrict__ in_Psi_Minus, hipDoubleComplex* __restrict__ in_n_Plus, hipDoubleComplex* __restrict__ in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k2_Psi_Plus, hipDoubleComplex* __restrict__ k2_Psi_Minus, hipDoubleComplex* __restrict__ k2_n_Plus, hipDoubleComplex* __restrict__ k2_n_Minus, hipDoubleComplex* __restrict__ k3_Psi_Plus, hipDoubleComplex* __restrict__ k3_Psi_Minus, hipDoubleComplex* __restrict__ k3_n_Plus, hipDoubleComplex* __restrict__ k3_n_Minus, hipDoubleComplex* __restrict__ k4_Psi_Plus, hipDoubleComplex* __restrict__ k4_Psi_Minus, hipDoubleComplex* __restrict__ k4_n_Plus, hipDoubleComplex* __restrict__ k4_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + RKCoefficients::b41 * dev_s_dt * k1_Psi_Plus[i] + RKCoefficients::b42 * dev_s_dt * k2_Psi_Plus[i] + RKCoefficients::b43 * dev_s_dt * k3_Psi_Plus[i] + RKCoefficients::b44 * dev_s_dt * k4_Psi_Plus[i];
        out_Psi_Minus[i] = in_Psi_Minus[i] + RKCoefficients::b41 * dev_s_dt * k1_Psi_Minus[i] + RKCoefficients::b42 * dev_s_dt * k2_Psi_Minus[i] + RKCoefficients::b43 * dev_s_dt * k3_Psi_Minus[i] + RKCoefficients::b44 * dev_s_dt * k4_Psi_Minus[i];
        out_n_Plus[i] = in_n_Plus[i] + RKCoefficients::b41 * dev_s_dt * k1_n_Plus[i] + RKCoefficients::b42 * dev_s_dt * k2_n_Plus[i] + RKCoefficients::b43 * dev_s_dt * k3_n_Plus[i] + RKCoefficients::b44 * dev_s_dt * k4_n_Plus[i];
        out_n_Minus[i] = in_n_Minus[i] + RKCoefficients::b41 * dev_s_dt * k1_n_Minus[i] + RKCoefficients::b42 * dev_s_dt * k2_n_Minus[i] + RKCoefficients::b43 * dev_s_dt * k3_n_Minus[i] + RKCoefficients::b44 * dev_s_dt * k4_n_Minus[i];
    }
}
/*
 * Summation of the k6 array
 */
__global__ void rungeFuncSumToK6( hipDoubleComplex* __restrict__ out_Psi_Plus, hipDoubleComplex* __restrict__ out_Psi_Minus, hipDoubleComplex* __restrict__ out_n_Plus, hipDoubleComplex* __restrict__ out_n_Minus, hipDoubleComplex* __restrict__ in_Psi_Plus, hipDoubleComplex* __restrict__ in_Psi_Minus, hipDoubleComplex* __restrict__ in_n_Plus, hipDoubleComplex* __restrict__ in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k2_Psi_Plus, hipDoubleComplex* __restrict__ k2_Psi_Minus, hipDoubleComplex* __restrict__ k2_n_Plus, hipDoubleComplex* __restrict__ k2_n_Minus, hipDoubleComplex* __restrict__ k3_Psi_Plus, hipDoubleComplex* __restrict__ k3_Psi_Minus, hipDoubleComplex* __restrict__ k3_n_Plus, hipDoubleComplex* __restrict__ k3_n_Minus, hipDoubleComplex* __restrict__ k4_Psi_Plus, hipDoubleComplex* __restrict__ k4_Psi_Minus, hipDoubleComplex* __restrict__ k4_n_Plus, hipDoubleComplex* __restrict__ k4_n_Minus, hipDoubleComplex* __restrict__ k5_Psi_Plus, hipDoubleComplex* __restrict__ k5_Psi_Minus, hipDoubleComplex* __restrict__ k5_n_Plus, hipDoubleComplex* __restrict__ k5_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + RKCoefficients::b51 * dev_s_dt * k1_Psi_Plus[i] + RKCoefficients::b52 * dev_s_dt * k2_Psi_Plus[i] + RKCoefficients::b53 * dev_s_dt * k3_Psi_Plus[i] + RKCoefficients::b54 * dev_s_dt * k4_Psi_Plus[i] + RKCoefficients::b55 * dev_s_dt * k5_Psi_Plus[i];
        out_Psi_Minus[i] = in_Psi_Minus[i] + RKCoefficients::b51 * dev_s_dt * k1_Psi_Minus[i] + RKCoefficients::b52 * dev_s_dt * k2_Psi_Minus[i] + RKCoefficients::b53 * dev_s_dt * k3_Psi_Minus[i] + RKCoefficients::b54 * dev_s_dt * k4_Psi_Minus[i] + RKCoefficients::b55 * dev_s_dt * k5_Psi_Minus[i];
        out_n_Plus[i] = in_n_Plus[i] + RKCoefficients::b51 * dev_s_dt * k1_n_Plus[i] + RKCoefficients::b52 * dev_s_dt * k2_n_Plus[i] + RKCoefficients::b53 * dev_s_dt * k3_n_Plus[i] + RKCoefficients::b54 * dev_s_dt * k4_n_Plus[i] + RKCoefficients::b55 * dev_s_dt * k5_n_Plus[i];
        out_n_Minus[i] = in_n_Minus[i] + RKCoefficients::b51 * dev_s_dt * k1_n_Minus[i] + RKCoefficients::b52 * dev_s_dt * k2_n_Minus[i] + RKCoefficients::b53 * dev_s_dt * k3_n_Minus[i] + RKCoefficients::b54 * dev_s_dt * k4_n_Minus[i] + RKCoefficients::b55 * dev_s_dt * k5_n_Minus[i];
    }
}

/**
 * Final Sum for the next iteration
 */
__global__ void rungeFuncSumToFinal( hipDoubleComplex* out_Psi_Plus, hipDoubleComplex* out_Psi_Minus, hipDoubleComplex* out_n_Plus, hipDoubleComplex* out_n_Minus, hipDoubleComplex* in_Psi_Plus, hipDoubleComplex* in_Psi_Minus, hipDoubleComplex* in_n_Plus, hipDoubleComplex* in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k3_Psi_Plus, hipDoubleComplex* __restrict__ k3_Psi_Minus, hipDoubleComplex* __restrict__ k3_n_Plus, hipDoubleComplex* __restrict__ k3_n_Minus, hipDoubleComplex* __restrict__ k4_Psi_Plus, hipDoubleComplex* __restrict__ k4_Psi_Minus, hipDoubleComplex* __restrict__ k4_n_Plus, hipDoubleComplex* __restrict__ k4_n_Minus, hipDoubleComplex* __restrict__ k5_Psi_Plus, hipDoubleComplex* __restrict__ k5_Psi_Minus, hipDoubleComplex* __restrict__ k5_n_Plus, hipDoubleComplex* __restrict__ k5_n_Minus, hipDoubleComplex* __restrict__ k6_Psi_Plus, hipDoubleComplex* __restrict__ k6_Psi_Minus, hipDoubleComplex* __restrict__ k6_n_Plus, hipDoubleComplex* __restrict__ k6_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + dev_s_dt * ( RKCoefficients::b61 * k1_Psi_Plus[i] + RKCoefficients::b63 * k3_Psi_Plus[i] + RKCoefficients::b64 * k4_Psi_Plus[i] + RKCoefficients::b65 * k5_Psi_Plus[i] + RKCoefficients::b66 * k6_Psi_Plus[i] );
        out_Psi_Minus[i] = in_Psi_Minus[i] + dev_s_dt * ( RKCoefficients::b61 * k1_Psi_Minus[i] + RKCoefficients::b63 * k3_Psi_Minus[i] + RKCoefficients::b64 * k4_Psi_Minus[i] + RKCoefficients::b65 * k5_Psi_Minus[i] + RKCoefficients::b66 * k6_Psi_Minus[i] );
        out_n_Plus[i] = in_n_Plus[i] + dev_s_dt * ( RKCoefficients::b61 * k1_n_Plus[i] + RKCoefficients::b63 * k3_n_Plus[i] + RKCoefficients::b64 * k4_n_Plus[i] + RKCoefficients::b65 * k5_n_Plus[i] + RKCoefficients::b66 * k6_n_Plus[i] );
        out_n_Minus[i] = in_n_Minus[i] + dev_s_dt * ( RKCoefficients::b61 * k1_n_Minus[i] + RKCoefficients::b63 * k3_n_Minus[i] + RKCoefficients::b64 * k4_n_Minus[i] + RKCoefficients::b65 * k5_n_Minus[i] + RKCoefficients::b66 * k6_n_Minus[i] );
    }
}
/**
 * Calculates the error sum for the RK45 Method
 * The error is calculated as the sum of the absolute values of the k arrays and is not normalized by either the sum of Psi nor the number of cells
 * @param out The output array
 */
__global__ void rungeFuncFinalError( hipDoubleComplex* out, hipDoubleComplex* in_Psi_Plus, hipDoubleComplex* in_Psi_Minus, hipDoubleComplex* in_n_Plus, hipDoubleComplex* in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k3_Psi_Plus, hipDoubleComplex* __restrict__ k3_Psi_Minus, hipDoubleComplex* __restrict__ k3_n_Plus, hipDoubleComplex* __restrict__ k3_n_Minus, hipDoubleComplex* __restrict__ k4_Psi_Plus, hipDoubleComplex* __restrict__ k4_Psi_Minus, hipDoubleComplex* __restrict__ k4_n_Plus, hipDoubleComplex* __restrict__ k4_n_Minus, hipDoubleComplex* __restrict__ k5_Psi_Plus, hipDoubleComplex* __restrict__ k5_Psi_Minus, hipDoubleComplex* __restrict__ k5_n_Plus, hipDoubleComplex* __restrict__ k5_n_Minus, hipDoubleComplex* __restrict__ k6_Psi_Plus, hipDoubleComplex* __restrict__ k6_Psi_Minus, hipDoubleComplex* __restrict__ k6_n_Plus, hipDoubleComplex* __restrict__ k6_n_Minus, hipDoubleComplex* __restrict__ k7_Psi_Plus, hipDoubleComplex* __restrict__ k7_Psi_Minus, hipDoubleComplex* __restrict__ k7_n_Plus, hipDoubleComplex* __restrict__ k7_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out[i] = make_hipDoubleComplex( 0.0, 0.0 );
        out[i] += abs2(dev_s_dt * ( RKCoefficients::e1 * k1_Psi_Plus[i] + RKCoefficients::e3 * k3_Psi_Plus[i] + RKCoefficients::e4 * k4_Psi_Plus[i] + RKCoefficients::e5 * k5_Psi_Plus[i] + RKCoefficients::e6 * k6_Psi_Plus[i] + RKCoefficients::e7 * k7_Psi_Plus[i] ));
        out[i] += abs2(dev_s_dt * ( RKCoefficients::e1 * k1_Psi_Minus[i] + RKCoefficients::e3 * k3_Psi_Minus[i] + RKCoefficients::e4 * k4_Psi_Minus[i] + RKCoefficients::e5 * k5_Psi_Minus[i] + RKCoefficients::e6 * k6_Psi_Minus[i] + RKCoefficients::e7 * k7_Psi_Minus[i] ));
        out[i] += abs2(dev_s_dt * ( RKCoefficients::e1 * k1_n_Plus[i] + RKCoefficients::e3 * k3_n_Plus[i] + RKCoefficients::e4 * k4_n_Plus[i] + RKCoefficients::e5 * k5_n_Plus[i] + RKCoefficients::e6 * k6_n_Plus[i] + RKCoefficients::e7 * k7_n_Plus[i] ));
        out[i] += abs2(dev_s_dt * ( RKCoefficients::e1 * k1_n_Minus[i] + RKCoefficients::e3 * k3_n_Minus[i] + RKCoefficients::e4 * k4_n_Minus[i] + RKCoefficients::e5 * k5_n_Minus[i] + RKCoefficients::e6 * k6_n_Minus[i] + RKCoefficients::e7 * k7_n_Minus[i] ));
    }
}


/**
 * Summation of the k2,k3 and k4 array
 * This function is used for the RK4
 */
__global__ void rungeFuncSum( double s_dt, hipDoubleComplex* __restrict__ out_Psi_Plus, hipDoubleComplex* __restrict__ out_Psi_Minus, hipDoubleComplex* __restrict__ out_n_Plus, hipDoubleComplex* __restrict__ out_n_Minus, hipDoubleComplex* __restrict__ in_Psi_Plus, hipDoubleComplex* __restrict__ in_Psi_Minus, hipDoubleComplex* __restrict__ in_n_Plus, hipDoubleComplex* __restrict__ in_n_Minus, hipDoubleComplex* __restrict__ k_Psi_Plus, hipDoubleComplex* __restrict__ k_Psi_Minus, hipDoubleComplex* __restrict__ k_n_Plus, hipDoubleComplex* __restrict__ k_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + s_dt * dev_s_dt * k_Psi_Plus[i];
        out_Psi_Minus[i] = in_Psi_Minus[i] + s_dt * dev_s_dt * k_Psi_Minus[i];
        out_n_Plus[i] = in_n_Plus[i] + s_dt * dev_s_dt * k_n_Plus[i];
        out_n_Minus[i] = in_n_Minus[i] + s_dt * dev_s_dt * k_n_Minus[i];
    }
}

__global__ void rungeFuncSumToFinalFixed( hipDoubleComplex* out_Psi_Plus, hipDoubleComplex* out_Psi_Minus, hipDoubleComplex* out_n_Plus, hipDoubleComplex* out_n_Minus, hipDoubleComplex* in_Psi_Plus, hipDoubleComplex* in_Psi_Minus, hipDoubleComplex* in_n_Plus, hipDoubleComplex* in_n_Minus, hipDoubleComplex* __restrict__ k1_Psi_Plus, hipDoubleComplex* __restrict__ k1_Psi_Minus, hipDoubleComplex* __restrict__ k1_n_Plus, hipDoubleComplex* __restrict__ k1_n_Minus, hipDoubleComplex* __restrict__ k2_Psi_Plus, hipDoubleComplex* __restrict__ k2_Psi_Minus, hipDoubleComplex* __restrict__ k2_n_Plus, hipDoubleComplex* __restrict__ k2_n_Minus, hipDoubleComplex* __restrict__ k3_Psi_Plus, hipDoubleComplex* __restrict__ k3_Psi_Minus, hipDoubleComplex* __restrict__ k3_n_Plus, hipDoubleComplex* __restrict__ k3_n_Minus, hipDoubleComplex* __restrict__ k4_Psi_Plus, hipDoubleComplex* __restrict__ k4_Psi_Minus, hipDoubleComplex* __restrict__ k4_n_Plus, hipDoubleComplex* __restrict__ k4_n_Minus ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int i = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( i < dev_s_N * dev_s_N ) {
        out_Psi_Plus[i] = in_Psi_Plus[i] + dev_s_dt / 6.0 * ( k1_Psi_Plus[i] + 2.0 * k2_Psi_Plus[i] + 2.0 * k3_Psi_Plus[i] + k4_Psi_Plus[i] );
        out_Psi_Minus[i] = in_Psi_Minus[i] + dev_s_dt / 6.0 * ( k1_Psi_Minus[i] + 2.0 * k2_Psi_Minus[i] + 2.0 * k3_Psi_Minus[i] + k4_Psi_Minus[i] );
        out_n_Plus[i] = in_n_Plus[i] + dev_s_dt / 6.0 * ( k1_n_Plus[i] + 2.0 * k2_n_Plus[i] + 2.0 * k3_n_Plus[i] + k4_n_Plus[i] );
        out_n_Minus[i] = in_n_Minus[i] + dev_s_dt / 6.0 * ( k1_n_Minus[i] + 2.0 * k2_n_Minus[i] + 2.0 * k3_n_Minus[i] + k4_n_Minus[i] );
    }
}