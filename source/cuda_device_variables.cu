#include "cuda_macro.cuh"
#include "cuda_complex.cuh"
#include "cuda_device_variables.cuh"

__device__ double dev_s_dx = 0;
__device__ double dev_p_g_r = 0;
__device__ int dev_s_N = 0;
__device__ double dev_p_m_eff = 0;
__device__ double dev_p_gamma_c = 0;
__device__ double dev_p_g_c = 0;
__device__ double dev_p_g_pm = 0;
__device__ double dev_p_gamma_r = 0;
__device__ double dev_p_R = 0;
__device__ double dev_p_delta_LT = 0;
__device__ double dev_s_dt = 0;
__device__ double dev_p_xmax = 0;
__device__ double dev_one_over_h_bar_s = 0;

__device__ int dev_n_pump = 0;
__device__ int dev_n_pulse = 0;

// Cached Device Variables
__device__ double dev_p_m_eff_scaled = 0;
__device__ double dev_p_delta_LT_scaled = 0;
__device__ hipDoubleComplex dev_pgr_plus_pR = {0,0};

// Pump and Pulse device arrays
double* dev_pump_amp = nullptr;
double* dev_pump_width = nullptr;
double* dev_pump_X = nullptr;
double* dev_pump_Y = nullptr;
int* dev_pump_pol = nullptr;
double* dev_pulse_t0 = nullptr;
double* dev_pulse_amp = nullptr;
double* dev_pulse_freq = nullptr;
double* dev_pulse_sigma = nullptr;
int* dev_pulse_m = nullptr;
int* dev_pulse_pol = nullptr;
double* dev_pulse_width = nullptr;
double* dev_pulse_X = nullptr;
double* dev_pulse_Y = nullptr;

// Device Pointers to input and output arrays
hipDoubleComplex* dev_current_Psi_Plus = nullptr;
hipDoubleComplex* dev_current_Psi_Minus = nullptr;
hipDoubleComplex* dev_current_n_Plus = nullptr;
hipDoubleComplex* dev_current_n_Minus = nullptr;
hipDoubleComplex* dev_next_Psi_Plus = nullptr;
hipDoubleComplex* dev_next_Psi_Minus = nullptr;
hipDoubleComplex* dev_next_n_Plus = nullptr;
hipDoubleComplex* dev_next_n_Minus = nullptr;

// Device Pointers to k1, k2, k3, k4, (k5, k6, k7) arrays
hipDoubleComplex* dev_k1_Psi_Plus = nullptr;
hipDoubleComplex* dev_k1_Psi_Minus = nullptr;
hipDoubleComplex* dev_k1_n_Plus = nullptr;
hipDoubleComplex* dev_k1_n_Minus = nullptr;
hipDoubleComplex* dev_k2_Psi_Plus = nullptr;
hipDoubleComplex* dev_k2_Psi_Minus = nullptr;
hipDoubleComplex* dev_k2_n_Plus = nullptr;
hipDoubleComplex* dev_k2_n_Minus = nullptr;
hipDoubleComplex* dev_k3_Psi_Plus = nullptr;
hipDoubleComplex* dev_k3_Psi_Minus = nullptr;
hipDoubleComplex* dev_k3_n_Plus = nullptr;
hipDoubleComplex* dev_k3_n_Minus = nullptr;
hipDoubleComplex* dev_k4_Psi_Plus = nullptr;
hipDoubleComplex* dev_k4_Psi_Minus = nullptr;
hipDoubleComplex* dev_k4_n_Plus = nullptr;
hipDoubleComplex* dev_k4_n_Minus = nullptr;
hipDoubleComplex* dev_k5_Psi_Plus = nullptr;
hipDoubleComplex* dev_k5_Psi_Minus = nullptr;
hipDoubleComplex* dev_k5_n_Plus = nullptr;
hipDoubleComplex* dev_k5_n_Minus = nullptr;
hipDoubleComplex* dev_k6_Psi_Plus = nullptr;
hipDoubleComplex* dev_k6_Psi_Minus = nullptr;
hipDoubleComplex* dev_k6_n_Plus = nullptr;
hipDoubleComplex* dev_k6_n_Minus = nullptr;
hipDoubleComplex* dev_k7_Psi_Plus = nullptr;
hipDoubleComplex* dev_k7_Psi_Minus = nullptr;
hipDoubleComplex* dev_k7_n_Plus = nullptr;
hipDoubleComplex* dev_k7_n_Minus = nullptr;

// Device Pointers to pulse and pulse2
hipDoubleComplex* dev_fft_plus = nullptr;
hipDoubleComplex* dev_fft_minus = nullptr;

double* dev_rk_error = nullptr;

// CUDA FFT Plan
hipfftHandle plan;

__device__ hipDoubleComplex dev_half_i = {0,0};
__device__ hipDoubleComplex dev_i = {0,0};
__device__ hipDoubleComplex dev_minus_half_i = {0,0};
__device__ hipDoubleComplex dev_minus_i = {0,0};

void initializeDeviceVariables( const double s_dx, const double s_dt, const double p_g_r, const int s_N, const double p_m_eff, const double p_gamma_c, const double p_g_c, const double p_g_pm, const double p_gamma_r, const double p_R, const double p_delta_LT, const double p_xmax, const double h_bar_s ) {
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_s_dx), &s_dx, sizeof( double ) ), "hipMemcpyToSymbol dx" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_s_dt), &s_dt, sizeof( double ) ), "hipMemcpyToSymbol dt" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_g_r), &p_g_r, sizeof( double ) ), "hipMemcpyToSymbol g_r" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_s_N), &s_N, sizeof( int ) ), "hipMemcpyToSymbol N" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_m_eff), &p_m_eff, sizeof( double ) ), "hipMemcpyToSymbol m_eff" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_gamma_c), &p_gamma_c, sizeof( double ) ), "hipMemcpyToSymbol gamma_c" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_g_c), &p_g_c, sizeof( double ) ), "hipMemcpyToSymbol g_c" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_g_pm), &p_g_pm, sizeof( double ) ), "hipMemcpyToSymbol g_pm" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_gamma_r), &p_gamma_r, sizeof( double ) ), "hipMemcpyToSymbol gamma_r" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_R), &p_R, sizeof( double ) ), "hipMemcpyToSymbol R" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_delta_LT), &p_delta_LT, sizeof( double ) ), "hipMemcpyToSymbol delta_LT" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_xmax), &p_xmax, sizeof( double ) ), "hipMemcpyToSymbol dev_p_xmax" );
    const auto one_over_h_bar_s = 1.0 / h_bar_s;
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_one_over_h_bar_s), &one_over_h_bar_s, sizeof( double ) ), "hipMemcpyToSymbol dev_one_over_h_bar_s" );
    // P/M 0.5i/1i
    auto half_i = make_hipDoubleComplex( 0.0, 0.5 );
    auto i = make_hipDoubleComplex( 0.0, 1.0 );
    auto minus_half_i = make_hipDoubleComplex( 0.0, -0.5 );
    auto minus_i = make_hipDoubleComplex( 0.0, -1.0 );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_half_i), &half_i, sizeof( hipDoubleComplex ) ), "hipMemcpyToSymbol dev_one_over_h_bar_s" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_i), &i, sizeof( hipDoubleComplex ) ), "hipMemcpyToSymbol dev_one_over_h_bar_s" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_minus_half_i), &minus_half_i, sizeof( hipDoubleComplex ) ), "hipMemcpyToSymbol dev_one_over_h_bar_s" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_minus_i), &minus_i, sizeof( hipDoubleComplex ) ), "hipMemcpyToSymbol dev_one_over_h_bar_s" );
    // Constant variables
    const auto p_m_eff_scaled = -0.5 / ( p_m_eff * s_dx * s_dx );
    const auto p_delta_LT_scaled = p_delta_LT / s_dx / s_dx;
    const auto pgr_plus_pR = make_hipDoubleComplex( p_g_r, 0.5 * p_R );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_m_eff_scaled), &p_m_eff_scaled, sizeof( double ) ), "hipMemcpyToSymbol dev_p_m_eff_scaled" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_p_delta_LT_scaled), &p_delta_LT_scaled, sizeof( double ) ), "hipMemcpyToSymbol dev_p_delta_LT_scaled" );
    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_pgr_plus_pR), &pgr_plus_pR, sizeof( hipDoubleComplex ) ), "hipMemcpyToSymbol dev_pgr_plus_pR" );
}

void initializePumpVariables( double* pump_amp, double* pump_width, double* pump_X, double* pump_Y, int* pump_pol, const int size ) {
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pump_amp, size * sizeof( double ) ), "malloc dev_pump_amp" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pump_amp, pump_amp, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pump_amp" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pump_width, size * sizeof( double ) ), "malloc dev_pump_width" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pump_width, pump_width, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pump_width" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pump_X, size * sizeof( double ) ), "malloc dev_pump_X" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pump_X, pump_X, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pump_X" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pump_Y, size * sizeof( double ) ), "malloc dev_pump_Y" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pump_Y, pump_Y, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pump_Y" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pump_pol, size * sizeof( int ) ), "malloc dev_pump_pol" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pump_pol, pump_pol, size * sizeof( int ), hipMemcpyHostToDevice ), "memcopy host to device dev_pump_pol" );

    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_n_pump), &size, sizeof( int ) ), "hipMemcpyToSymbol dev_n_pump" );
}

void initializePulseVariables( double* pulse_t0, double* pulse_amp, double* pulse_freq, double* pulse_sigma, int* pulse_m, int* pulse_pol, double* pulse_width, double* pulse_X, double* pulse_Y, const int size ) {
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_t0, size * sizeof( double ) ), "malloc dev_pulse_t0" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_t0, pulse_t0, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_t0" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_amp, size * sizeof( double ) ), "malloc dev_pulse_amp" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_amp, pulse_amp, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_amp" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_freq, size * sizeof( double ) ), "malloc dev_pulse_freq" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_freq, pulse_freq, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_freq" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_sigma, size * sizeof( double ) ), "malloc dev_pulse_sigma" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_sigma, pulse_sigma, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_sigma" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_m, size * sizeof( int ) ), "malloc dev_pulse_m" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_m, pulse_m, size * sizeof( int ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_m" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_pol, size * sizeof( int ) ), "malloc dev_pulse_pol" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_pol, pulse_pol, size * sizeof( int ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_pol" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_width, size * sizeof( double ) ), "malloc dev_pulse_width" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_width, pulse_width, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_width" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_X, size * sizeof( double ) ), "malloc dev_pulse_X" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_X, pulse_X, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_X" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_pulse_Y, size * sizeof( double ) ), "malloc dev_pulse_Y" );
    CHECK_CUDA_ERROR( hipMemcpy( dev_pulse_Y, pulse_Y, size * sizeof( double ), hipMemcpyHostToDevice ), "memcopy host to device dev_pulse_Y" );

    CHECK_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_n_pulse), &size, sizeof( int ) ), "hipMemcpyToSymbol dev_n_pulse" );
}

/**
 * Initialize device arrays to zero
 */
void initializeDeviceArrays( const int s_N ) {
    const auto size = s_N * s_N;
    // std::unique_ptr dummy = std::make_unique<hipDoubleComplex[]>( size );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_current_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_current_Psi_Plus" )
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_current_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_current_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_current_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_current_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_current_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_current_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_next_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_next_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_next_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_next_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_next_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_next_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_next_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_next_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k1_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k1_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k1_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k1_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k1_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k1_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k1_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k1_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k2_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k2_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k2_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k2_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k2_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k2_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k2_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k2_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k3_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k3_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k3_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k3_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k3_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k3_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k3_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k3_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k4_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k4_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k4_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k4_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k5_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k5_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k5_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k5_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k6_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k6_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k6_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k6_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k7_Psi_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k7_Psi_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_Psi_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k7_n_Plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_k7_n_Minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_k4_n_Minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_fft_plus, size * sizeof( hipDoubleComplex ) ), "malloc dev_fft_plus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_fft_minus, size * sizeof( hipDoubleComplex ) ), "malloc dev_fft_minus" );
    CHECK_CUDA_ERROR( hipMalloc( (void**)&dev_rk_error, size * sizeof( double ) ), "malloc dev_rk_error" );

    CHECK_CUDA_ERROR( hipfftPlan2d( &plan, s_N, s_N, HIPFFT_Z2Z ), "FFT Plan" );
}

void setDeviceArrays( Scalar* psi_plus, Scalar* psi_minus, Scalar* n_plus, Scalar* n_minus, const int s_N ) {
    const auto size = s_N * s_N;
    if ( psi_plus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( dev_current_Psi_Plus, psi_plus, size * sizeof( hipDoubleComplex ), hipMemcpyHostToDevice ), "memcopy host to device psi_plus" );
    if ( psi_minus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( dev_current_Psi_Minus, psi_minus, size * sizeof( hipDoubleComplex ), hipMemcpyHostToDevice ), "memcopy host to device psi_minus" );
    if ( n_plus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( dev_current_n_Plus, n_plus, size * sizeof( hipDoubleComplex ), hipMemcpyHostToDevice ), "memcopy host to device n_plus" );
    if ( n_minus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( dev_current_n_Minus, n_minus, size * sizeof( hipDoubleComplex ), hipMemcpyHostToDevice ), "memcopy host to device n_minus" );
}

void getDeviceArrays( Scalar* psi_plus, Scalar* psi_minus, Scalar* n_plus, Scalar* n_minus, Scalar* fft_plus, Scalar* fft_minus, const int s_N ) {
    const auto size = s_N * s_N;
    if ( psi_plus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( psi_plus, dev_current_Psi_Plus, size * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host psi_plus" );
    if ( psi_minus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( psi_minus, dev_current_Psi_Minus, size * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host psi_minus" );
    if ( n_plus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( n_plus, dev_current_n_Plus, size * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host n_plus" );
    if ( n_minus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( n_minus, dev_current_n_Minus, size * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host n_minus" );
    if ( fft_plus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( fft_plus, dev_fft_plus, size * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host fft_plus" );
    if ( fft_minus != nullptr )
        CHECK_CUDA_ERROR( hipMemcpy( fft_minus, dev_fft_minus, size * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host fft_minus" );
}

void getDeviceArraySlice(Scalar* buffer_in, Scalar* buffer_out, const int start, const int length) {
    CHECK_CUDA_ERROR( hipMemcpy( buffer_out, buffer_in + start, length * sizeof( hipDoubleComplex ), hipMemcpyDeviceToHost ), "memcpy device to host buffer" );
}

void freeDeviceArrays() {
    for ( const auto pointer : { dev_current_Psi_Plus, dev_current_Psi_Minus, dev_current_n_Plus, dev_current_n_Minus, dev_next_Psi_Plus, dev_next_Psi_Minus, dev_next_n_Plus, dev_next_n_Minus, dev_k1_Psi_Plus, dev_k1_Psi_Minus, dev_k1_n_Plus, dev_k1_n_Minus, dev_k2_Psi_Plus, dev_k2_Psi_Minus, dev_k2_n_Plus, dev_k2_n_Minus, dev_k3_Psi_Plus, dev_k3_Psi_Minus, dev_k3_n_Plus, dev_k3_n_Minus, dev_k4_Psi_Plus, dev_k4_Psi_Minus, dev_k4_n_Plus, dev_k4_n_Minus, dev_k5_Psi_Plus, dev_k5_Psi_Minus, dev_k5_n_Plus, dev_k5_n_Minus, dev_k6_Psi_Plus, dev_k6_Psi_Minus, dev_k6_n_Plus, dev_k6_n_Minus, dev_k7_Psi_Plus, dev_k7_Psi_Minus, dev_k7_n_Plus, dev_k7_n_Minus, dev_fft_plus, dev_fft_minus } ) {
        CHECK_CUDA_ERROR( hipFree( pointer ), "free" );
    }
    for ( const auto pointer : { dev_pump_amp, dev_pump_width, dev_pump_X, dev_pump_Y, dev_pulse_t0, dev_pulse_amp, dev_pulse_freq, dev_pulse_sigma, dev_pulse_width, dev_pulse_X, dev_pulse_Y } ) {
        CHECK_CUDA_ERROR( hipFree( pointer ), "free" );
    }
    for ( const auto pointer : { dev_pump_pol, dev_pulse_m, dev_pulse_pol } ) {
        CHECK_CUDA_ERROR( hipFree( pointer ), "free" );
    }
    CHECK_CUDA_ERROR( hipFree( dev_rk_error ), "free" );
    CHECK_CUDA_ERROR( hipfftDestroy( plan ), "FFT Destroy" );
}