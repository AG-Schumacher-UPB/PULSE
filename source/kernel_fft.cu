#include "hip/hip_runtime.h"
#include "cuda_macro.cuh"
#include <hip/hip_runtime.h>
#include "cuda_complex_math.cuh"
#include "kernel_fft.cuh"

__global__ void kernel_makeFFTVisible( hipDoubleComplex* input, hipDoubleComplex* output ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int index = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( index < dev_s_N * dev_s_N ) {
        const auto val = input[index];
        output[index] = make_hipDoubleComplex( log( val.x * val.x + val.y * val.y ), 0 );
    }
}

__global__ void fftshift_2D( hipDoubleComplex* data_plus, hipDoubleComplex* data_minus, int N_half ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int index = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( index >= dev_s_N * dev_s_N )
        return;
    // Current indices of upper left quadrant
    const int i = device_floor( index / dev_s_N );
    if ( i >= N_half )
        return;
    const int j = index % dev_s_N;
    if ( j >= N_half )
        return;
    // Swap upper left quadrant with lower right quadrant
    swap_symbol( data_plus[i * dev_s_N + j], data_plus[( i + N_half ) * dev_s_N + j + N_half] );
    swap_symbol( data_minus[i * dev_s_N + j], data_minus[( i + N_half ) * dev_s_N + j + N_half] );
    // Swap lower left quadrant with upper right quadrant
    swap_symbol( data_plus[i * dev_s_N + j + N_half], data_plus[( i + N_half ) * dev_s_N + j] );
    swap_symbol( data_minus[i * dev_s_N + j + N_half], data_minus[( i + N_half ) * dev_s_N + j] );
}

__global__ void kernel_maskFFT( hipDoubleComplex* data_plus, hipDoubleComplex* data_minus, const double s, const double w, bool out_mask ) {
    int blockId = ( gridDim.x * blockIdx.y ) + blockIdx.x;
    int index = ( blockId * ( blockDim.x * blockDim.y ) ) + ( threadIdx.y * blockDim.x ) + threadIdx.x;
    if ( index < dev_s_N * dev_s_N ) {
        const int i = device_floor( index / dev_s_N );
        const int j = index % dev_s_N;
        double ky = 2. * i / dev_s_N - 1.;
        double kx = 2. * j / dev_s_N - 1.;
        double mask = exp( -1.0 * pow( ( kx * kx + ky * ky ) / w / w, s ) );
        data_plus[index] = out_mask ? make_hipDoubleComplex( sqrt( mask ), 0 ) : data_plus[index] / dev_s_N / dev_s_N * mask;
        data_minus[index] = out_mask ? make_hipDoubleComplex( sqrt( mask ), 0 ) : data_minus[index] / dev_s_N / dev_s_N * mask;
    }
}