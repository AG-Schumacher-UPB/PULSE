#include "kernel/kernel_hamilton.cuh"

CUDA_DEVICE void PC3::Hamilton::scalar( complex_number& regular, complex_number* __restrict__ vector, int index, const int row, const int col, const int N, const bool periodic ) {
    complex_number upper, lower, left, right;
    if (periodic) {
        upper = upper_neighbour_periodic( vector, index, row, col, 1, N );
        lower = lower_neighbour_periodic( vector, index, row, col, 1, N );
        left = left_neighbour_periodic( vector, index, row, col, 1, N );
        right = right_neighbour_periodic( vector, index, row, col, 1, N );
    } else {
        upper = upper_neighbour( vector, index, row, col, 1, N );
        lower = lower_neighbour( vector, index, row, col, 1, N );
        left = left_neighbour( vector, index, row, col, 1, N );
        right = right_neighbour( vector, index, row, col, 1, N );
    }
    regular = -4.0 * vector[index] + upper + lower + left + right;
}

CUDA_DEVICE void PC3::Hamilton::tetm_plus( complex_number& regular, complex_number& cross, complex_number* __restrict__ vector, int index, const int row, const int col, const int N, const bool periodic ) {
    complex_number upper, lower, left, right;
    if (periodic) {
        upper = upper_neighbour_periodic( vector, index, row, col, 1, N );
        lower = lower_neighbour_periodic( vector, index, row, col, 1, N );
        left = left_neighbour_periodic( vector, index, row, col, 1, N );
        right = right_neighbour_periodic( vector, index, row, col, 1, N );
        cross = upper + lower - left - right + complex_number(0.0,-0.5) * ( right_neighbour_periodic( vector, index - N, row - 1, col, 1, N ) - left_neighbour_periodic( vector, index - N, row - 1, col, 1, N ) - right_neighbour_periodic( vector, index + N, row + 1, col, 1, N ) + left_neighbour_periodic( vector, index + N, row + 1, col, 1, N ) );
    } else {
        upper = upper_neighbour( vector, index, row, col, 1, N );
        lower = lower_neighbour( vector, index, row, col, 1, N );
        left = left_neighbour( vector, index, row, col, 1, N );
        right = right_neighbour( vector, index, row, col, 1, N );
        cross = upper + lower - left - right + complex_number(0.0,-0.5) * ( right_neighbour( vector, index - N, row - 1, col, 1, N ) - left_neighbour( vector, index - N, row - 1, col, 1, N ) - right_neighbour( vector, index + N, row + 1, col, 1, N ) + left_neighbour( vector, index + N, row + 1, col, 1, N ) );
    }
    regular = -4.0 * vector[index] + upper + lower + left + right;
}

CUDA_DEVICE void PC3::Hamilton::tetm_minus( complex_number& regular, complex_number& cross, complex_number* __restrict__ vector, int index, const int row, const int col, const int N, const bool periodic ) {
    complex_number upper, lower, left, right;
    if (periodic) {
        upper = upper_neighbour_periodic( vector, index, row, col, 1, N );
        lower = lower_neighbour_periodic( vector, index, row, col, 1, N );
        left = left_neighbour_periodic( vector, index, row, col, 1, N );
        right = right_neighbour_periodic( vector, index, row, col, 1, N );
        cross = upper + lower - left - right + complex_number(0.0,0.5) * ( right_neighbour_periodic( vector, index - N, row - 1, col, 1, N ) - left_neighbour_periodic( vector, index - N, row - 1, col, 1, N ) - right_neighbour_periodic( vector, index + N, row + 1, col, 1, N ) + left_neighbour_periodic( vector, index + N, row + 1, col, 1, N ) );
    } else {
        upper = upper_neighbour( vector, index, row, col, 1, N );
        lower = lower_neighbour( vector, index, row, col, 1, N );
        left = left_neighbour( vector, index, row, col, 1, N );
        right = right_neighbour( vector, index, row, col, 1, N );
        cross = upper + lower - left - right + complex_number(0.0,0.5) * ( right_neighbour( vector, index - N, row - 1, col, 1, N ) - left_neighbour( vector, index - N, row - 1, col, 1, N ) - right_neighbour( vector, index + N, row + 1, col, 1, N ) + left_neighbour( vector, index + N, row + 1, col, 1, N ) );
    }
    regular = -4. * vector[index] + upper + lower + left + right;
}