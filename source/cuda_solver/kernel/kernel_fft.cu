#include "hip/hip_runtime.h"
#include "cuda/cuda_macro.cuh"
#include "cuda/cuda_complex.cuh"
#include "kernel/kernel_index_overwrite.cuh"
#include "kernel/kernel_fft.cuh"
#ifndef USECPU
#include <hip/hip_runtime.h>
#endif

CUDA_GLOBAL void kernel_make_fft_visible( int i, complex_number* input, complex_number* output, const unsigned int N ) {
    GET_THREAD_INDEX( i, N );
    
    const auto val = input[i];
    output[i] = { PC3::CUDA::log( PC3::CUDA::real(val) * PC3::CUDA::real(val) + PC3::CUDA::imag(val) * PC3::CUDA::imag(val) ), 0 };
}

CUDA_GLOBAL void fft_shift_2D( int i, complex_number* data, const unsigned int N_x, const unsigned int N_y ) {
    GET_THREAD_INDEX( i, N_x*N_y );

    // Current indices of upper left quadrant
    const int k = i / N_x ;
    if ( k >= N_y/2 )
        return;
    const int l = i % N_x;
    if ( l >= N_x/2 )
        return;

    // Swap upper left quadrant with lower right quadrant
    swap_symbol( data[k * N_x + l], data[( k + N_y/2 ) * N_x + l + N_x/2] );
    
    // Swap lower left quadrant with upper right quadrant
    swap_symbol( data[k * N_x + l + N_x/2], data[( k + N_y/2 ) * N_x + l] );
}

CUDA_GLOBAL void kernel_mask_fft( int i, complex_number* data, real_number* mask, const unsigned int N ) {
    GET_THREAD_INDEX( i, N );

    data[i] = data[i] / real_number(N) * mask[i];
}