#include "hip/hip_runtime.h"
#include "kernel/kernel_runge_function.cuh"
#include "kernel/kernel_hamilton.cuh"
#include "kernel/kernel_index_overwrite.cuh"

PULSE_GLOBAL void PC3::Kernel::Compute::gp_tetm( int i, Type::real t, MatrixContainer::Pointers dev_ptrs, SystemParameters::KernelParameters p_in, Solver::Oscillation::Pointers oscillation_pulse, Solver::Oscillation::Pointers oscillation_pump, Solver::Oscillation::Pointers oscillation_potential, InputOutput io ) {
    
    LOCAL_SHARE_STRUCT( SystemParameters::KernelParameters, p_in, p );
    
    OVERWRITE_THREAD_INDEX( i );

    const int row = i / p.N_x;
    const int col = i % p.N_x;

    const auto in_wf_plus = io.in_wf_plus[i];
    const auto in_wf_minus = io.in_wf_minus[i];

    Type::complex hamilton_regular_plus = p.m2_over_dx2_p_dy2 * in_wf_plus;
    Type::complex hamilton_regular_minus = p.m2_over_dx2_p_dy2 * in_wf_minus;
    Type::complex hamilton_cross_plus, hamilton_cross_minus;
    PC3::Kernel::Hamilton::tetm_plus( hamilton_regular_plus, hamilton_cross_minus, io.in_wf_plus, i, row, col, p.N_x, p.N_y, p.dx, p.dy, p.periodic_boundary_x, p.periodic_boundary_y );
    PC3::Kernel::Hamilton::tetm_minus( hamilton_regular_minus, hamilton_cross_plus, io.in_wf_minus, i, row, col, p.N_x, p.N_y, p.dx, p.dy, p.periodic_boundary_x, p.periodic_boundary_y );

    const auto in_rv_plus = io.in_rv_plus[i];
    const auto in_rv_minus = io.in_rv_minus[i];
    const Type::real in_psi_plus_norm = CUDA::abs2( in_wf_plus );
    const Type::real in_psi_minus_norm = CUDA::abs2( in_wf_minus );
 
    // MARK: Wavefunction Plus
    Type::complex result = p.minus_i_over_h_bar_s * p.m_eff_scaled * hamilton_regular_plus;
    
    for (int k = 0; k < oscillation_potential.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const Type::complex potential = dev_ptrs.potential_plus[i+offset] * CUDA::gaussian_oscillator(t, oscillation_potential.t0[k], oscillation_potential.sigma[k], oscillation_potential.freq[k]);
        result += p.minus_i_over_h_bar_s * potential * in_wf_plus;
    }

    result += p.minus_i_over_h_bar_s * p.g_c * in_psi_plus_norm * in_wf_plus;
    result += p.minus_i_over_h_bar_s * p.g_r * in_rv_plus * in_wf_plus;
    result += Type::real(0.5) * p.R * in_rv_plus * in_wf_plus;
    result -= Type::real(0.5)* p.gamma_c * in_wf_plus;

    result += p.minus_i_over_h_bar_s * p.g_pm * in_psi_minus_norm * in_wf_plus;
    result += p.minus_i_over_h_bar_s * p.delta_LT * hamilton_cross_plus;
    
    // MARK: Pulse Plus
    for (int k = 0; k < oscillation_pulse.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const Type::complex pulse = dev_ptrs.pulse_plus[i+offset];
        result += p.one_over_h_bar_s * pulse * CUDA::gaussian_complex_oscillator(t, oscillation_pulse.t0[k], oscillation_pulse.sigma[k], oscillation_pulse.freq[k]);
    }

    // MARK: Stochastic
    if (p.stochastic_amplitude > 0.0) {
        const Type::complex dw = dev_ptrs.random_number[i] * CUDA::sqrt( ( p.R * in_rv_plus + p.gamma_c ) / (Type::real(4.0) * p.dV) );
        result -= p.minus_i_over_h_bar_s * p.g_c * in_wf_plus / p.dV - dw / p.dt;
    }

    io.out_wf_plus[i] = result;

    // MARK: Reservoir Plus
    result = -( p.gamma_r + p.R * in_psi_plus_norm ) * in_rv_plus;

    for (int k = 0; k < oscillation_pump.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const auto gauss = CUDA::gaussian_oscillator(t, oscillation_pump.t0[k], oscillation_pump.sigma[k], oscillation_pump.freq[k]);
        result += dev_ptrs.pump_plus[i+offset] * gauss;
    }

    // MARK: Stochastic-2
    if (p.stochastic_amplitude > 0.0)
        result += p.R * in_rv_plus / p.dV;

    io.out_rv_plus[i] = result;
    

    // MARK: Wavefunction Minus
    result = p.minus_i_over_h_bar_s * p.m_eff_scaled * hamilton_regular_minus;
    
    for (int k = 0; k < oscillation_potential.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const Type::complex potential = dev_ptrs.potential_minus[i+offset] * CUDA::gaussian_oscillator(t, oscillation_potential.t0[k], oscillation_potential.sigma[k], oscillation_potential.freq[k]);
        result += p.minus_i_over_h_bar_s * potential * in_wf_minus;
    }

    result += p.minus_i_over_h_bar_s * p.g_c * in_psi_minus_norm * in_wf_minus;
    result += p.minus_i_over_h_bar_s * p.g_r * in_rv_minus * in_wf_minus;
    result += Type::real(0.5) * p.R * in_rv_minus * in_wf_minus;
    result -= Type::real(0.5) * p.gamma_c * in_wf_minus;
 
    result += p.minus_i_over_h_bar_s * p.g_pm * in_psi_plus_norm * in_wf_minus;
    result += p.minus_i_over_h_bar_s * p.delta_LT * hamilton_cross_minus;

    // MARK: Pulse Minus
    for (int k = 0; k < oscillation_pulse.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const Type::complex pulse = dev_ptrs.pulse_minus[i+offset];
        result += p.one_over_h_bar_s * pulse * CUDA::gaussian_complex_oscillator(t, oscillation_pulse.t0[k], oscillation_pulse.sigma[k], oscillation_pulse.freq[k]);
    }

    if (p.stochastic_amplitude > 0.0) {
        const Type::complex dw = dev_ptrs.random_number[i] * CUDA::sqrt( ( p.R * in_rv_minus + p.gamma_c ) / (Type::real(4.0) * p.dV) );
        result -= p.minus_i_over_h_bar_s * p.g_c * in_wf_minus / p.dV - dw / p.dt;
    }

    io.out_wf_minus[i] = result;

    // MARK: Reservoir Minus
    result = -( p.gamma_r + p.R * in_psi_minus_norm ) * in_rv_minus;

    for (int k = 0; k < oscillation_pump.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const auto gauss = CUDA::gaussian_oscillator(t, oscillation_pump.t0[k], oscillation_pump.sigma[k], oscillation_pump.freq[k]);
        result += dev_ptrs.pump_minus[i+offset] * gauss;
    }

    // MARK: Stochastic-2
    if (p.stochastic_amplitude > 0.0)
        result += p.R * in_rv_minus / p.dV;

    io.out_rv_minus[i] = result;

}