#include "hip/hip_runtime.h"
#include "kernel/kernel_runge_function.cuh"
#include "kernel/kernel_hamilton.cuh"
#include "kernel/kernel_index_overwrite.cuh"

/**
 * Mode without TE/TM Splitting
 * The differential equation for this model reduces to
 * ...
 */
CUDA_GLOBAL void PC3::Kernel::Compute::gp_scalar( int i, real_number t, MatrixContainer::Pointers dev_ptrs, System::Parameters p, Solver::Oscillation::Pointers oscillation_pulse, Solver::Oscillation::Pointers oscillation_pump, Solver::Oscillation::Pointers oscillation_potential, InputOutput io ) {
    
    OVERWRITE_THREAD_INDEX( i );

    complex_number hamilton;
    PC3::Hamilton::scalar( hamilton, io.in_wf_plus, i, i / p.N_x /*Row*/, i % p.N_x /*Col*/, p.N_x, p.N_y, p.dx, p.dy, p.periodic_boundary_x, p.periodic_boundary_y );
    
    const complex_number in_wf = io.in_wf_plus[i];
    const complex_number in_rv = io.in_rv_plus[i];
    const real_number in_psi_norm = CUDA::abs2( in_wf );
    
    // MARK: Wavefunction
    complex_number result = p.minus_i_over_h_bar_s * ( p.m_eff_scaled * hamilton );

    for (int k = 0; k < oscillation_potential.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const complex_number potential = dev_ptrs.potential_plus[i+offset] * PC3::CUDA::gaussian_oscillator(t, oscillation_potential.t0[k], oscillation_potential.sigma[k], oscillation_potential.freq[k]);
        result += p.minus_i_over_h_bar_s * potential * in_wf;
    }
    result += p.minus_i_over_h_bar_s * in_wf * dev_ptrs.pulse_plus[i];

    result += p.minus_i_over_h_bar_s * p.g_c * in_psi_norm * in_wf;
    result += p.minus_i_over_h_bar_s * p.g_r * in_rv * in_wf;
    result += real_number(0.5) * p.R * in_rv * in_wf;
    result -= real_number(0.5) * p.gamma_c * in_wf;

    // MARK: Pulse
    for (int k = 0; k < oscillation_pulse.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const complex_number pulse = dev_ptrs.pulse_plus[i+offset];
        result += p.minus_i_over_h_bar_s * pulse * PC3::CUDA::gaussian_complex_oscillator(t, oscillation_pulse.t0[k], oscillation_pulse.sigma[k], oscillation_pulse.freq[k]);
    }
    result += p.minus_i_over_h_bar_s * dev_ptrs.pulse_plus[i];
    // MARK: Stochastic
    if (p.stochastic_amplitude > 0.0) {
        const complex_number dw = dev_ptrs.random_number[i] * PC3::CUDA::sqrt( ( p.R * in_rv + p.gamma_c ) / (real_number(4.0) * p.dV) );
        result -= p.minus_i_over_h_bar_s * p.g_c * in_wf / p.dV - dw / p.dt;
    }
    
    io.out_wf_plus[i] = result;
    
    // MARK: Reservoir
    result = -p.gamma_r * in_rv;
    result -= p.R * in_psi_norm * in_rv;
    for (int k = 0; k < oscillation_pump.n; k++) {
        const int offset = k * p.N_x * p.N_y;
        result += dev_ptrs.pump_plus[i+offset] * PC3::CUDA::gaussian_oscillator(t, oscillation_pump.t0[k], oscillation_pump.sigma[k], oscillation_pump.freq[k]);
    }
    result += dev_ptrs.pump_plus[i];

    // MARK: Stochastic-2
    if (p.stochastic_amplitude > 0.0)
        result += p.R * in_rv / p.dV;
    io.out_rv_plus[i] = result;

}

/**
 * Linear, Nonlinear and Independet parts of the upper Kernel
 * These isolated implementations serve for the Split Step
 * Fourier Method (SSFM)
*/

CUDA_GLOBAL void PC3::Kernel::Compute::gp_scalar_linear_fourier( int i, real_number t, MatrixContainer::Pointers dev_ptrs, System::Parameters p, Solver::Oscillation::Pointers oscillation_pulse, Solver::Oscillation::Pointers oscillation_pump, Solver::Oscillation::Pointers oscillation_potential, InputOutput io ) {
    
    OVERWRITE_THREAD_INDEX( i );
    size_t row = i / p.N_x;
    size_t col = i % p.N_x;
    
    real_number k_x = 3.1415926535 * (col-p.N_x/2)/p.L_x;
    real_number k_y = 3.1415926535 * (row-p.N_y/2)/p.L_y;

    real_number linear = p.m_eff_scaled * (k_x*k_x + k_y*k_y);
    io.out_wf_plus[i] = io.in_wf_plus[i] / real_number(p.N2) * CUDA::exp( p.one_over_h_bar_s * linear * p.dt / 2.0 );
}

CUDA_GLOBAL void PC3::Kernel::Compute::gp_scalar_nonlinear( int i, real_number t, MatrixContainer::Pointers dev_ptrs, System::Parameters p, Solver::Oscillation::Pointers oscillation_pulse, Solver::Oscillation::Pointers oscillation_pump, Solver::Oscillation::Pointers oscillation_potential, InputOutput io ) {
    
    OVERWRITE_THREAD_INDEX( i );
    
    const complex_number in_wf = io.in_wf_plus[i];
    
    const real_number in_psi_norm = CUDA::abs2( in_wf );
    
    // MARK: Wavefunction
    complex_number result = p.minus_i_over_h_bar_s * p.g_c * in_psi_norm * in_wf;
    result -= real_number(0.5) * p.gamma_c;

    for (int k = 0; k < oscillation_potential.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const complex_number potential = dev_ptrs.potential_plus[i+offset] * PC3::CUDA::gaussian_oscillator(t, oscillation_potential.t0[k], oscillation_potential.sigma[k], oscillation_potential.freq[k]);
        result += p.minus_i_over_h_bar_s * potential;
    }

    result = in_wf * CUDA::exp(result * p.dt);

    // MARK: Pulse
    for (int k = 0; k < oscillation_pulse.n; k++) {
        const size_t offset = k * p.N_x * p.N_y;
        const complex_number pulse = dev_ptrs.pulse_plus[i+offset];
        result += p.minus_i_over_h_bar_s * pulse * PC3::CUDA::gaussian_complex_oscillator(t, oscillation_pulse.t0[k], oscillation_pulse.sigma[k], oscillation_pulse.freq[k]);
    }
    result += p.minus_i_over_h_bar_s * dev_ptrs.pulse_plus[i] * p.dt;

    io.out_wf_plus[i] = result;
    
}
