#include "hip/hip_runtime.h"
#include "cuda/cuda_complex.cuh"
#include "cuda/cuda_macro.cuh"
#include "solver/gpu_solver.hpp"
#include "kernel/kernel_normalize_imaginary_time_propagation.cuh"
#include "misc/helperfunctions.hpp"
#include <iostream>
void PC3::Solver::normalizeImaginaryTimePropagation( MatrixContainer::Pointers device_pointers, System::Parameters p, dim3 block_size, dim3 grid_size ) {
    if (not system.imaginary_time)
        return;
        
    // Calculate min and max values
    auto [minimum_plus, maximum_plus] = CUDA::minmax( device_pointers.buffer_wavefunction_plus, system.p.N2, true /*is a device pointer*/ );
    auto [minimum_plus_r, maximum_plus_r] = CUDA::minmax( device_pointers.buffer_reservoir_plus, system.p.N2, true /*is a device pointer*/ );
    real_number maximum_minus, maximum_minus_r;
    if (system.p.use_twin_mode) {
        real_number dummy;
        std::tie(dummy, maximum_minus) = CUDA::minmax( device_pointers.buffer_wavefunction_minus, system.p.N2, true /*is a device pointer*/ );
        std::tie(dummy, maximum_minus_r) = CUDA::minmax( device_pointers.buffer_reservoir_minus, system.p.N2, true /*is a device pointer*/ );
    }
    
    complex_number wf =  {CUDA::abs2(maximum_plus), CUDA::abs2(maximum_minus)};
    complex_number rv =  {CUDA::abs2(maximum_plus_r), CUDA::abs2(maximum_minus_r)};

    CALL_KERNEL(
        Kernel::normalize_imaginary_time_propagation, "Imag Time", grid_size, block_size,
        device_pointers, p, wf, rv
    );
    
}