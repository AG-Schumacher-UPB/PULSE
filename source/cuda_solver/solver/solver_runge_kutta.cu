#include "hip/hip_runtime.h"
#ifndef USECPU
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>
#else
#include <numeric>
#endif

#include <complex>
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/cuda_complex.cuh"
#include "cuda/cuda_macro.cuh"
#include "kernel/kernel_runge_function.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_fft.cuh"
#include "system/system.hpp"
#include "misc/helperfunctions.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "kernel/kernel_random_numbers.cuh"

/*
 * Helper variable for caching the current time for FFT evaluations.
 * We dont need this variable anywhere else, so we just create it
 * locally to this file here.
 */
real_number cached_t = 0.0;

// Helper macro to choose the correct runge function
#define RUNGE_FUNCTION_GP (system.use_twin_mode ? PC3::Kernel::Compute::gp_tetm : PC3::Kernel::Compute::gp_scalar)
#define RUNGE_FUNCTION_RE (system.use_twin_mode ? PC3::Kernel::Compute::tetm_reservoir : PC3::Kernel::Compute::scalar_reservoir)
#define RUNGE_FUNCTION_PULSE (system.use_twin_mode ? PC3::Kernel::Compute::tetm_pulse : PC3::Kernel::Compute::scalar_pulse)
#define RUNGE_FUNCTION_STOCHASTIC (system.use_twin_mode ? PC3::Kernel::Compute::tetm_stochastic : PC3::Kernel::Compute::scalar_stochastic)

// Helper Macro to iterate a specific RK K
#define CALCULATE_K( index, time, input_wavefunction, input_reservoir ) \
CALL_KERNEL( \
    RUNGE_FUNCTION_GP, "K"#index, grid_size, block_size,  \
    time, device_pointers, p, \
    {  \
        device_pointers.input_wavefunction##_plus, device_pointers.input_wavefunction##_minus, device_pointers.input_reservoir##_plus, device_pointers.input_reservoir##_minus, \
        device_pointers.k##index##_wavefunction_plus, device_pointers.k##index##_wavefunction_minus, device_pointers.k##index##_reservoir_plus, device_pointers.k##index##_reservoir_minus \
    } \
); \
if (evaluate_reservoir) \
    CALL_KERNEL( \
        RUNGE_FUNCTION_RE, "K"#index"_Reservoir", grid_size, block_size, \
        time, device_pointers, p, \
        {  \
            device_pointers.input_wavefunction##_plus, device_pointers.input_wavefunction##_minus, device_pointers.input_reservoir##_plus, device_pointers.input_reservoir##_minus, \
            device_pointers.k##index##_wavefunction_plus, device_pointers.k##index##_wavefunction_minus, device_pointers.k##index##_reservoir_plus, device_pointers.k##index##_reservoir_minus \
        } \
    ); \
if (evaluate_pulse) \
    CALL_KERNEL( \
        RUNGE_FUNCTION_PULSE, "K"#index"_Pulse", grid_size, block_size, \
        time, device_pointers, p, pulse_pointers, \
        {  \
            device_pointers.input_wavefunction##_plus, device_pointers.input_wavefunction##_minus, device_pointers.input_reservoir##_plus, device_pointers.input_reservoir##_minus, \
            device_pointers.k##index##_wavefunction_plus, device_pointers.k##index##_wavefunction_minus, device_pointers.k##index##_reservoir_plus, device_pointers.k##index##_reservoir_minus \
        } \
    ); \
if (evaluate_stochastic) \
    CALL_KERNEL( \
        RUNGE_FUNCTION_STOCHASTIC, "K"#index"_Stochastic", grid_size, block_size, \
        time, device_pointers, p, \
        {  \
            device_pointers.input_wavefunction##_plus, device_pointers.input_wavefunction##_minus, device_pointers.input_reservoir##_plus, device_pointers.input_reservoir##_minus, \
            device_pointers.k##index##_wavefunction_plus, device_pointers.k##index##_wavefunction_minus, device_pointers.k##index##_reservoir_plus, device_pointers.k##index##_reservoir_minus \
        } \
    ); \


/*
 * This function iterates the Runge Kutta Kernel using a fixed time step.
 * A 4th order Runge-Kutta method is used. This function calls a single
 * rungeFuncSum function with varying delta-t. Calculation of the inputs
 * for the next rungeFuncKernel call is done in the rungeFuncSum function.
 * The general implementation of the RK4 method goes as follows:
 * ------------------------------------------------------------------------------
 * k1 = f(t, y) = rungeFuncKernel(current)
 * input_for_k2 = current + 0.5 * dt * k1
 * k2 = f(t + 0.5 * dt, input_for_k2) = rungeFuncKernel(input_for_k2)
 * input_for_k3 = current + 0.5 * dt * k2
 * k3 = f(t + 0.5 * dt, input_for_k3) = rungeFuncKernel(input_for_k3)
 * input_for_k4 = current + dt * k3
 * k4 = f(t + dt, input_for_k4) = rungeFuncKernel(input_for_k4)
 * next = current + dt * (1/6 * k1 + 1/3 * k2 + 1/3 * k3 + 1/6 * k4)
 * ------------------------------------------------------------------------------
 * @param evaluate_pulse If true, the pulse is evaluated at the current time step
 * ------------------------------------------------------------------------------
 * The Runge method iterates psi,k1-k4 to psi_next using a wave-like approach.
 * We calculate 4 rows of k1, 3 rows of k2, 2 rows of k3 and 1 row of k4 before the first iteration.
 * Then, we iterate all of the remaining rows after each other, incrementing the buffer for the next iteration.
 */

void PC3::Solver::iterateFixedTimestepRungeKutta( dim3 block_size, dim3 grid_size ) {
    // This variable contains all the system parameters the kernel could need
    auto p = system.snapshotParameters();
    
    // This variable contains all the device pointers the kernel could need
    auto device_pointers = device.pointers();

    // The CPU should briefly evaluate wether the pulse and the reservoir have to be evaluated
    bool evaluate_pulse = system.evaluatePulse();
    bool evaluate_reservoir = system.evaluateReservoir();
    bool evaluate_stochastic = system.evaluateStochastic();

    // Pointers to Pulse Variables. This is subject to change
    auto pulse_pointers = dev_pulse_oscillation.pointers();

    // The delta time is either real or imaginary, depending on the system configuration
    complex_number delta_time = system.imaginary_time ? complex_number(0.0, -system.dt) : complex_number(system.dt, 0.0);

    // If required, calculate new set of random numbers.
    if (evaluate_stochastic)
    CALL_KERNEL(
        PC3::Kernel::generate_random_numbers, "random_number_gen", grid_size, block_size,
        device_pointers.random_state, device_pointers.random_number, system.s_N_x*system.s_N_y, system.stochastic_amplitude*PC3::CUDA::sqrt(system.dt), system.stochastic_amplitude*PC3::CUDA::sqrt(system.dt)
    );

    CALCULATE_K( 1, system.t, wavefunction, reservoir );

    CALL_KERNEL(
        Kernel::RK4::runge_sum_to_input_k2, "Sum for K2", grid_size, block_size,
        delta_time, device_pointers, p, system.use_twin_mode
    );

    CALCULATE_K( 2, system.t + 0.5 * system.dt, buffer_wavefunction, buffer_reservoir );

    CALL_KERNEL(
        Kernel::RK4::runge_sum_to_input_k3, "Sum for K3", grid_size, block_size,
        delta_time, device_pointers, p, system.use_twin_mode
    );

    CALCULATE_K( 3, system.t + 0.5 * system.dt, buffer_wavefunction, buffer_reservoir);

    CALL_KERNEL(
        Kernel::RK4::runge_sum_to_input_k4, "Sum for K4", grid_size, block_size,
        delta_time, device_pointers, p, system.use_twin_mode
    );

    CALCULATE_K( 4, system.t + system.dt, buffer_wavefunction, buffer_reservoir);

    CALL_KERNEL(
        Kernel::RK4::runge_sum_to_final, "Final Sum", grid_size, block_size,
        delta_time, device_pointers, p, system.use_twin_mode
    );

    CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    device.wavefunction_plus.swap( device.buffer_wavefunction_plus );
    device.reservoir_plus.swap( device.buffer_reservoir_plus );
    if ( system.use_twin_mode ) {
        device.wavefunction_minus.swap( device.buffer_wavefunction_minus );
        device.reservoir_minus.swap( device.buffer_reservoir_minus );
    }

    return;
}

struct square_reduction
{
    CUDA_HOST_DEVICE real_number operator()(const complex_number& x) const { 
        const real_number res = PC3::CUDA::abs2(x);
        return res; 
    }
};


/*
* This function iterates the Runge Kutta Kernel using a variable time step.
* A 4th order Runge-Kutta method is used to calculate
* the next y iteration; a 5th order solution is
* used to calculate the iteration error.
* This function calls multiple different rungeFuncSum functions with varying
* delta-t and coefficients. Calculation of the inputs for the next
* rungeFuncKernel call is done in the rungeFuncSum function.
* The general implementation of the RK45 method goes as follows:
* ------------------------------------------------------------------------------
* k1 = f(t, y) = rungeFuncKernel(current)
* input_for_k2 = current + b11 * dt * k1
* k2 = f(t + a2 * dt, input_for_k2) = rungeFuncKernel(input_for_k2)
* input_for_k3 = current + b21 * dt * k1 + b22 * dt * k2
* k3 = f(t + a3 * dt, input_for_k3) = rungeFuncKernel(input_for_k3)
* input_for_k4 = current + b31 * dt * k1 + b32 * dt * k2 + b33 * dt * k3
* k4 = f(t + a4 * dt, input_for_k4) = rungeFuncKernel(input_for_k4)
* input_for_k5 = current + b41 * dt * k1 + b42 * dt * k2 + b43 * dt * k3
                 + b44 * dt * k4
* k5 = f(t + a5 * dt, input_for_k5) = rungeFuncKernel(input_for_k5)
* input_for_k6 = current + b51 * dt * k1 + b52 * dt * k2 + b53 * dt * k3
                 + b54 * dt * k4 + b55 * dt * k5
* k6 = f(t + a6 * dt, input_for_k6) = rungeFuncKernel(input_for_k6)
* next = current + dt * (b61 * k1 + b63 * k3 + b64 * k4 + b65 * k5 + b66 * k6)
* k7 = f(t + a7 * dt, next) = rungeFuncKernel(next)
* error = dt * (e1 * k1 + e3 * k3 + e4 * k4 + e5 * k5 + e6 * k6 + e7 * k7)
* ------------------------------------------------------------------------------
* The error is then used to update the timestep; If the error is below threshold,
* the iteration is accepted and the total time is increased by dt. If the error
* is above threshold, the iteration is rejected and the timestep is decreased.
* The timestep is always bounded by dt_min and dt_max and will only increase
* using whole multiples of dt_min.
* ------------------------------------------------------------------------------
* @param system The system to iterate
* @param evaluate_pulse If true, the pulse is evaluated at the current time step
*/
void PC3::Solver::iterateVariableTimestepRungeKutta( dim3 block_size, dim3 grid_size ) {
    // Accept current step?
    bool accept = false;
    // This variable contains all the device pointers the kernel could need
    auto device_pointers = device.pointers();
    // This variable contains all the system parameters the kernel could need
    // Pointers to Pulse Variables. This is subject to change
    auto pulse_pointers = dev_pulse_oscillation.pointers();
    // The CPU should briefly evaluate wether the pulse and the reservoir have to be evaluated
    bool evaluate_pulse = system.evaluatePulse();
    bool evaluate_reservoir = system.evaluateReservoir();
    bool evaluate_stochastic = system.evaluateStochastic();

    // The delta time is either real or imaginary, depending on the system configuration
    complex_number delta_time = system.imaginary_time ? complex_number(0.0, -system.dt) : complex_number(system.dt, 0.0);

    // If required, calculate new set of random numbers.
    if (evaluate_stochastic)
    CALL_KERNEL(
        PC3::Kernel::generate_random_numbers, "random_number_gen", grid_size, block_size,
        device_pointers.random_state, device_pointers.random_number, system.s_N_x*system.s_N_y, system.stochastic_amplitude*PC3::CUDA::sqrt(system.dt), system.stochastic_amplitude*PC3::CUDA::sqrt(system.dt)
    );

    do {
        // We snapshot here to make sure that the dt is updated
        auto p = system.snapshotParameters();

        CALCULATE_K( 1, system.t, wavefunction, reservoir );

        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_to_input_of_k2, "Sum for K2", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );

        CALCULATE_K( 2, system.t + RKCoefficients::a2 * system.dt, buffer_wavefunction, buffer_reservoir );

        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_to_input_of_k3, "Sum for K3", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );


        CALCULATE_K( 3, system.t + RKCoefficients::a3 * system.dt, buffer_wavefunction, buffer_reservoir );

        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_to_input_of_k4, "Sum for K4", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );

        CALCULATE_K( 4, system.t + RKCoefficients::a4 * system.dt, buffer_wavefunction, buffer_reservoir );

        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_to_input_of_k5, "Sum for K5", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );

        CALCULATE_K( 5, system.t + RKCoefficients::a5 * system.dt, buffer_wavefunction, buffer_reservoir );

        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_to_input_of_k6, "Sum for K6", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );

        CALCULATE_K( 6, system.t + RKCoefficients::a6 * system.dt, buffer_wavefunction, buffer_reservoir );

        // Final Result is in the buffer_ arrays
        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_to_final, "Final Sum", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );

        CALCULATE_K( 7, system.t + RKCoefficients::a7 * system.dt, buffer_wavefunction, buffer_reservoir );

        CALL_KERNEL(
            PC3::Kernel::RK45::runge_sum_final_error, "Final Sum Error", grid_size, block_size, 
            delta_time, device_pointers, p, system.use_twin_mode
        );

        #ifndef USECPU
        real_number final_error = thrust::reduce( THRUST_DEVICE, device.rk_error.get(), device.rk_error.get() + system.s_N_x * system.s_N_y, 0.0, thrust::plus<real_number>() );
        real_number sum_abs2 = thrust::transform_reduce( THRUST_DEVICE, device.wavefunction_plus.get(), device.wavefunction_plus.get() + system.s_N_x * system.s_N_y, square_reduction(), 0.0, thrust::plus<real_number>() );
        #else
        real_number final_error = std::reduce( device.rk_error.get(), device.rk_error.get() + system.s_N_x * system.s_N_y, 0.0, std::plus<real_number>() );
        real_number sum_abs2 = std::transform_reduce( device.wavefunction_plus.get(), device.wavefunction_plus.get() + system.s_N_x * system.s_N_y, 0.0, std::plus<real_number>(), square_reduction() );
        #endif

        // TODO: maybe go back to using max since thats faster
        //auto plus_max = std::get<1>( minmax( device.wavefunction_plus.get(), system.s_N_x * system.s_N_y, true /*Device Pointer*/ ) );
        final_error = final_error / sum_abs2;

        // Calculate dh
        real_number dh = std::pow( system.tolerance / 2. / CUDA::max( final_error, 1E-15 ), 0.25 );
        // Check if dh is nan
        if ( std::isnan( dh ) ) {
            dh = 1.0;
        }
        if ( std::isnan( final_error ) )
            dh = 0.5;
        
        //  Set new timestep
        // system.dt = min(system.dt * dh, system.dt_max);
        //if ( dh < 1.0 )
        //    system.dt = CUDA::max( system.dt - system.dt_min * CUDA::floor( 1.0 / dh ), system.dt_min );
        //    //system.dt -= system.dt_min;
        //else
        //    system.dt = CUDA::min( system.dt + system.dt_min * CUDA::floor( dh ), system.dt_max );
        //    //system.dt += system.dt_min;
//
        //// Make sure to also update dt from p
        //p.dt = system.dt;
final_error = 0;
        // Accept step if error is below tolerance
        if ( final_error < system.tolerance ) {
            accept = true;
            // Since the "next" Y is in the buffer_ arrays, we swap current_wavefunction and buffer_wf
            // This is fast, because we just swap pointers instead of copying data.
            device.wavefunction_plus.swap( device.buffer_wavefunction_plus );
            device.reservoir_plus.swap( device.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                device.wavefunction_minus.swap( device.buffer_wavefunction_minus );
                device.reservoir_minus.swap( device.buffer_reservoir_minus );
            }
        }
    } while ( !accept );
}

/*
 * This function calculates the Fast Fourier Transformation of Psi+ and Psi-
 * and saves the result in dev_fft_plus and dev_fft_minus. These values can
 * then be grabbed using the getDeviceArrays() function. The FFT is then
 * shifted such that k = 0 is in the center of the FFT matrix. Then, the
 * FFT Filter is applied to the FFT, and the FFT is shifted back. Finally,
 * the inverse FFT is calculated and the result is saved in dev_current_Psi_Plus
 * and dev_current_Psi_Minus. The FFT Arrays are shifted once again for
 * visualization purposes.
 * NOTE/OPTIMIZATION: The Shift->Filter->Shift function will be changed later
 * to a cached filter mask, which itself will be shifted.
 */
void PC3::Solver::applyFFTFilter( dim3 block_size, dim3 grid_size, bool apply_mask ) {
    #ifndef USECPU
    // Calculate the actual FFTs
    CHECK_CUDA_ERROR( FFTSOLVER( plan, (fft_complex_number*)device.wavefunction_plus.get(), (fft_complex_number*)device.fft_plus.get(), HIPFFT_FORWARD ), "FFT Exec" );


    // For now, we shift, transform, shift the results. TODO: Move this into one function without shifting
    // Shift FFT to center k = 0
    fft_shift_2D<<<grid_size, block_size>>>( device.fft_plus.get(), system.s_N_x, system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Shift Plus" );

    // Do the FFT and the shifting here already for visualization only
    if ( system.use_twin_mode ) {
        CHECK_CUDA_ERROR( FFTSOLVER( plan, (fft_complex_number*)device.wavefunction_minus.get(), (fft_complex_number*)device.fft_minus.get(), HIPFFT_FORWARD ), "FFT Exec" );
        fft_shift_2D<<<grid_size, block_size>>>( device.fft_minus.get(), system.s_N_x, system.s_N_y );
        CHECK_CUDA_ERROR( {}, "FFT Shift Minus" );
    }
    
    if (not apply_mask)
        return;
    
    // Apply the FFT Mask Filter
    kernel_mask_fft<<<grid_size, block_size>>>( device.fft_plus.get(), device.fft_mask_plus.get(), system.s_N_x*system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Filter" )
    
    // Undo the shift
    fft_shift_2D<<<grid_size, block_size>>>( device.fft_plus.get(), system.s_N_x, system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Shift" )

    // Transform back.
    CHECK_CUDA_ERROR( FFTSOLVER( plan, device.fft_plus.get(), device.wavefunction_plus.get(), HIPFFT_BACKWARD ), "iFFT Exec" );
    
    // Shift FFT Once again for visualization
    fft_shift_2D<<<grid_size, block_size>>>( device.fft_plus.get(), system.s_N_x, system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Shift" );
    
    // Do the same for the minus component
    if (not system.use_twin_mode)
        return;
    kernel_mask_fft<<<grid_size, block_size>>>( device.fft_minus.get(), device.fft_mask_minus.get(), system.s_N_x*system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Filter" )
    fft_shift_2D<<<grid_size, block_size>>>( device.fft_minus.get(), system.s_N_x,system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Shift" )
    CHECK_CUDA_ERROR( FFTSOLVER( plan, device.fft_minus.get(), device.wavefunction_minus.get(), HIPFFT_BACKWARD ), "iFFT Exec" );
    fft_shift_2D<<<grid_size, block_size>>>( device.fft_minus.get(), system.s_N_x,system.s_N_y );
    CHECK_CUDA_ERROR( {}, "FFT Shift" );
    #endif
}

bool first_time = true;

/**
 * Iterates the Runge-Kutta-Method on the GPU
 * Note, that all device arrays and variables have to be initialized at this point
 * @param t Current time, will be updated to t + dt
 * @param dt Time step, will be updated to the next time step
 * @param s_N_x Number of grid points in one dimension
 * @param s_N_y Number of grid points in the other dimension
 */
bool PC3::Solver::iterateRungeKutta( ) {


    // First, check if the maximum time has been reached
    if ( system.t >= system.t_max )
        return false;

    dim3 block_size( system.block_size, 1 );
    dim3 grid_size( ( system.s_N_x*system.s_N_y + block_size.x ) / block_size.x, 1 );
    
    if (first_time and system.evaluateStochastic()) {
        first_time = false;
        auto device_pointers = device.pointers();
        CALL_KERNEL(
                PC3::Kernel::initialize_random_number_generator, "random_number_init", grid_size, block_size,
                system.random_seed, device_pointers.random_state, system.s_N_x*system.s_N_y
            );
        std::cout << "Initialized Random Number Generator" << std::endl;
    }
    
    if ( system.fixed_time_step )
        iterateFixedTimestepRungeKutta( block_size, grid_size );
    else
        iterateVariableTimestepRungeKutta( block_size, grid_size );

    // Syncronize
    //CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
    
    // Increase t.
    system.t = system.t + system.dt;

    // For statistical purposes, increase the iteration counter
    system.iteration++;

    // FFT Guard 
    if ( system.t - cached_t < system.fft_every )
        return true;

    // Calculate the FFT
    cached_t = system.t; 
    applyFFTFilter( block_size, grid_size, system.fft_mask.size() > 0 );

    return true;
    // Syncronize
    //CHECK_CUDA_ERROR( hipDeviceSynchronize(), "Sync" );
}